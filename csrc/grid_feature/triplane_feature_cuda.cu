#include "hip/hip_runtime.h"
// Copyright 2022 Sony Group Corporation.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <pybind11/pybind11.h>

#include <pybind11/stl.h>
#include <hip/hip_vector_types.h>
#include <cuda_common.cuh>
#include <grid_feature/common.cuh>
#include <grid_feature/common_triplane.cuh>

namespace py = pybind11;

using namespace grid_feature;
using namespace grid_feature::triplane;

/***
    Forward
 ***/


namespace triplane_feature {

//TODO: optimize access pattern like voxel hash

__global__
void kernel_query_on_triplane(int N, float *output, const float *query, const float *feature,
                                 int G, int D, 
                                 float3 min, float3 max, 
                                 bool boundary_check) {
                                 
  auto stride_u = G * D;
  auto stride_v = D;
  auto G1 = G - 1.f;
  auto grid_sizes1 = make_float3(G1, G1, G1);
  
  NBLA_CUDA_KERNEL_LOOP(n, N) {  // N = (B * R * N) * (D * 3)
    auto n_idx = flat_to_3d(n, D);
    auto b = n_idx.x, d = n_idx.y, i = n_idx.z;

    auto querys = *(float3*)(query + b * 3);
  
    // continuous point
    auto scales = grid_sizes1 / (max - min);
    auto xyz = (querys - min) * scales;

    // discrete points
    auto xyz0 = floorf(xyz);
    xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
    xyz0 = fminf(xyz0, grid_sizes1);
    auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);
    
    // coefficients
    auto pqr0 = xyz1 - xyz;
    auto pqr1 = 1.f - pqr0;

    // grid features
    auto feature_index = [&](const uint u, const uint v) {
      return (u * stride_u) + (v * stride_v) + d;
    };

    auto locs = select_locations(i, xyz0, xyz1);
    auto coef = select_coefficients(i, pqr0, pqr1);
    auto u0 = locs.x, u1 = locs.y, v0 = locs.z, v1 = locs.w;
    auto a0 = coef.x, a1 = coef.y, b0 = coef.z, b1 = coef.w;

    auto feature_i = feature + i * (G * G * D);
    auto f00 = feature_i[feature_index(u0, v0)];
    auto f01 = feature_i[feature_index(u0, v1)];
    auto f10 = feature_i[feature_index(u1, v0)];
    auto f11 = feature_i[feature_index(u1, v1)];
    
    // linear interpolation
    auto f = a0 * b0 * f00 + a0 * b1 * f01 + a1 * b0 * f10 + a1 * b1 * f11;
    output[n] = f;
  }
}


void query_on_triplane(int N, int64_t output_ptr, int64_t query_ptr, int64_t feature_ptr, 
                          int G, int D, 
                          std::vector<float> min, std::vector<float> max, 
                          bool boundary_check) {
  auto output_buff = reinterpret_cast<float*>(output_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);
  auto feature_buff = reinterpret_cast<float*>(feature_ptr);

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_query_on_triplane, N, 
                                 output_buff, query_buff, feature_buff, 
                                 G, D, 
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}


/***
    Backward (1st-order)
    1. wrt query
    2. wrt feature
 ***/

__global__
void kernel_grad_query(int N, float *grad_query, 
                       const float *grad_output, 
                       const float *query, const float *feature,
                       int G, int D, 
                       float3 min, float3 max, 
                       bool boundary_check) {
  auto stride_u = G * D;
  auto stride_v = D;
  auto G1 = G - 1.f;
  auto grid_sizes1 = make_float3(G1, G1, G1);

  NBLA_CUDA_KERNEL_LOOP(n, N) {  // N = (B * R * N) * (D * 3)
    auto n_idx = flat_to_3d(n, D);
    auto b = n_idx.x, d = n_idx.y, i = n_idx.z;

    auto querys = *(float3*)(query + b * 3);

     // continuous point   
    auto scales = grid_sizes1 / (max - min);
    auto xyz = (querys - min) * scales;

    // discrete points
    auto xyz0 = floorf(xyz);
    xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
    xyz0 = fminf(xyz0, grid_sizes1);
    auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);

    // coefficients
    auto pqr0 = xyz1 - xyz;
    auto pqr1 = 1.f - pqr0;

    // grid features
    auto feature_index = [&](const uint u, const uint v) {
      return (u * stride_u) + (v * stride_v) + d;
    };

    auto locs = select_locations(i, xyz0, xyz1);
    auto coef = select_coefficients(i, pqr0, pqr1);
    auto u0 = locs.x, u1 = locs.y, v0 = locs.z, v1 = locs.w;
    auto a0 = coef.x, a1 = coef.y, b0 = coef.z, b1 = coef.w;

    auto feature_i = feature + i * (G * G * D);
    auto f00 = feature_i[feature_index(u0, v0)];
    auto f01 = feature_i[feature_index(u0, v1)];
    auto f10 = feature_i[feature_index(u1, v0)];
    auto f11 = feature_i[feature_index(u1, v1)];

    // gradients
    auto ograd = grad_output[n];
    auto compute_grad = [&](float scale, float c0, float c1, float d0, float d1) {
      return ograd * scale * (c0 * d0 + c1 * d1);
    };
    auto scale = select_scales(i, scales);
    auto su = scale.x, sv = scale.y;
    auto gu = compute_grad(su, b0, b1, (f10 - f00), (f11 - f01));
    auto gv = compute_grad(sv, a0, a1, (f01 - f00), (f11 - f10));

    auto shift = select_shift(i);
    atomicAdd(grad_query + b * 3 + shift.x, gu);
    atomicAdd(grad_query + b * 3 + shift.y, gv);
  }
}

void grad_query(int N, int64_t grad_query_ptr, 
                int64_t grad_output_ptr, int64_t query_ptr, int64_t feature_ptr, 
                int G, int D, 
                std::vector<float> min, std::vector<float> max, 
                bool boundary_check, bool accum) {
  auto grad_query_buff = reinterpret_cast<float*>(grad_query_ptr);
  auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);
  auto feature_buff = reinterpret_cast<float*>(feature_ptr);

  if (!accum) {
    auto size = N / D;
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_zero, size, grad_query_buff);
  }
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_query, N, 
                                 grad_query_buff, 
                                 grad_output_buff, query_buff, feature_buff, 
                                 G, D, 
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}


__global__
void kernel_grad_feature(int N, float *grad_feature, 
                         const float *grad_output, 
                         const float *query, 
                         int G, int D, 
                         float3 min, float3 max, 
                         bool boundary_check) {
  auto stride_u = G * D;
  auto stride_v = D;
  auto G1 = G - 1.f;
  auto grid_sizes1 = make_float3(G1, G1, G1);

  NBLA_CUDA_KERNEL_LOOP(n, N) {  // N = (B * R * N) * (D * 3)
    auto n_idx = flat_to_3d(n, D);
    auto b = n_idx.x, d = n_idx.y, i = n_idx.z;

    auto querys = *(float3*)(query + b * 3);  
    
    // continuous point   
    auto scales = grid_sizes1 / (max - min);
    auto xyz = (querys - min) * scales;

    // discrete points
    auto xyz0 = floorf(xyz);
    xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
    xyz0 = fminf(xyz0, grid_sizes1);
    auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);

    // coefficients
    auto pqr0 = xyz1 - xyz;
    auto pqr1 = 1.f - pqr0;
    
    // gradients
    auto ograd = grad_output[n];
    auto grad_feature_i = grad_feature + i * (G * G * D);
    auto compute_grad = [&](const uint u, const uint v, 
                            const float a, const float b) {
      auto f_idx = (u * stride_u) + (v * stride_v) + d;
      atomicAdd(grad_feature_i + f_idx, ograd * a * b);
    };

    auto locs = select_locations(i, xyz0, xyz1);
    auto coef = select_coefficients(i, pqr0, pqr1);
    auto u0 = locs.x, u1 = locs.y, v0 = locs.z, v1 = locs.w;
    auto a0 = coef.x, a1 = coef.y, b0 = coef.z, b1 = coef.w;

    compute_grad(u0, v0, a0, b0);
    compute_grad(u0, v1, a0, b1);
    compute_grad(u1, v0, a1, b0);
    compute_grad(u1, v1, a1, b1);
    
  }
}


void grad_feature(int N, int64_t grad_feature_ptr, 
                  int64_t grad_output_ptr, int64_t query_ptr, 
                  int G, int D, 
                  std::vector<float> min, std::vector<float> max, 
                  bool boundary_check, bool accum) {
  auto grad_feature_buff = reinterpret_cast<float*>(grad_feature_ptr);
  auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);

  
  if (!accum) {
    auto size = 3 * G * G * D;
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_zero, size, grad_feature_buff);
  }

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_feature, N, 
                                 grad_feature_buff, 
                                 grad_output_buff, query_buff, 
                                 G, D, 
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}


/***
    Backward (2nd-order)
    1-1. grad_query wrt grad_output
    // 1-2. grad_query wrt query
    1-3. grad_query wrt feature
    // 2-1. grad_feature wrt grad_output
    // 2-2. grad_feature wrt query

    naming rule, kernel_<backward_function_name>_grad_<wrt_input>
    
 ***/


// 1-1. grad_query wrt grad_output
template<bool accum = false>
__global__
void kernel_grad_query_grad_grad_output(int N, float *grad_grad_output, 
                                        const float *grad_grad_query, 
                                        const float *query, const float *feature,
                                        int G, int D, 
                                        float3 min, float3 max, 
                                        bool boundary_check) {
  auto stride_u = G * D;
  auto stride_v = D;
  auto G1 = G - 1.f;
  auto grid_sizes1 = make_float3(G1, G1, G1);

  NBLA_CUDA_KERNEL_LOOP(n, N) {  // N = (B * R * N) * (D * 3)
    auto n_idx = flat_to_3d(n, D);
    auto b = n_idx.x, d = n_idx.y, i = n_idx.z;

    auto querys = *(float3*)(query + b * 3);
  
    // continuous point   
    auto scales = grid_sizes1 / (max - min);
    auto xyz = (querys - min) * scales;

    // discrete points
    auto xyz0 = floorf(xyz);
    xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
    xyz0 = fminf(xyz0, grid_sizes1);
    auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);

    // coefficients
    auto pqr0 = xyz1 - xyz;
    auto pqr1 = 1.f - pqr0;

    // grid features
    auto feature_index = [&](const uint u, const uint v) {
      return (u * stride_u) + (v * stride_v) + d;
    };
    
    auto locs = select_locations(i, xyz0, xyz1);
    auto coef = select_coefficients(i, pqr0, pqr1);
    auto u0 = locs.x, u1 = locs.y, v0 = locs.z, v1 = locs.w;
    auto a0 = coef.x, a1 = coef.y, b0 = coef.z, b1 = coef.w;

    auto feature_i = feature + i * (G * G * D);
    auto f00 = feature_i[feature_index(u0, v0)];
    auto f01 = feature_i[feature_index(u0, v1)];
    auto f10 = feature_i[feature_index(u1, v0)];
    auto f11 = feature_i[feature_index(u1, v1)];

    // gradients
    auto compute_grad_term = [&](float ggt, float scale, float c0, float c1, float d0, float d1) {
      return ggt * scale * (c0 * d0 + c1 * d1);
    };
    auto gg_xyz = *(float3*)(grad_grad_query + b * 3);
    auto gguv = select_gguv(i, gg_xyz);
    auto ggu = gguv.x, ggv = gguv.y;
    auto scale = select_scales(i, scales);
    auto su = scale.x, sv = scale.y;
    
    auto ggo_u = compute_grad_term(ggu, su, b0, b1, (f10 - f00), (f11 - f01));
    auto ggo_v = compute_grad_term(ggv, sv, a0, a1, (f01 - f00), (f11 - f10));
    auto ggo = ggo_u + ggo_v;
    grad_grad_output[n] = accum ? grad_grad_output[n] + ggo : ggo;
  }
}


void grad_query_grad_grad_output(int N, int64_t grad_grad_output_ptr, 
                                 int64_t grad_grad_query_ptr, 
                                 int64_t query_ptr, int64_t feature_ptr, 
                                 int G, int D, 
                                 std::vector<float> min, std::vector<float> max, 
                                 bool boundary_check, bool accum) {
  auto grad_grad_output_buff = reinterpret_cast<float*>(grad_grad_output_ptr);
  auto grad_grad_query_buff = reinterpret_cast<float*>(grad_grad_query_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);
  auto feature_buff = reinterpret_cast<float*>(feature_ptr);

  auto kernel = accum 
    ? kernel_grad_query_grad_grad_output<true> 
    : kernel_grad_query_grad_grad_output<false>;
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel, N, 
                                 grad_grad_output_buff, 
                                 grad_grad_query_buff, 
                                 query_buff, feature_buff, 
                                 G, D, 
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}

// // 1-2. grad_query wrt query
// __global__
// void kernel_grad_query_grad_query(int N, float *grad_query, 
//                                   const float *grad_grad_query, 
//                                   const float *grad_output, 
//                                   const float *query, const float *feature,
//                                   int G, int D, 
//                                   float3 min, float3 max, 
//                                   bool boundary_check) {
//   auto Gy0 = grid_sizes.y;
//   auto Gz0 = grid_sizes.z;
//   auto stride_x = Gy0 * Gz0 * D;
//   auto stride_y = Gz0 * D;
//   auto stride_z = D;
//   auto grid_sizes1 = to_float3(grid_sizes) - 1.f;
  
//   NBLA_CUDA_KERNEL_LOOP(n, N) {

//     auto b = n / D;
//     auto d = n - b * D;

//     auto querys = *(float3*)(query + b * 3);
  
//     // continuous point   
//     auto scales = grid_sizes1 / (max - min);
//     auto xyz = (querys - min) * scales;

//     // discrete points
//     auto xyz0 = floorf(xyz);    
//     xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
//     xyz0 = fminf(xyz0, grid_sizes1);
//     auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);

//     // coefficients
//     auto pqr0 = xyz1 - xyz;
//     auto pqr1 = 1.f - pqr0;

//     // scalars
//     uint x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;
//     uint x1 = xyz1.x, y1 = xyz1.y, z1 = xyz1.z;
//     auto p0 = pqr0.x, q0 = pqr0.y, r0 = pqr0.z;
//     auto p1 = pqr1.x, q1 = pqr1.y, r1 = pqr1.z;
//     auto sx = scales.x, sy = scales.y, sz = scales.z;

//     // grid features
//     auto feature_index = [&](const uint x, const uint y, const uint z) {
//       return x * stride_x + y * stride_y + z * stride_z + d;
//     };
    
//     auto f000 = feature[feature_index(x0, y0, z0)];
//     auto f001 = feature[feature_index(x0, y0, z1)];
//     auto f010 = feature[feature_index(x0, y1, z0)];
//     auto f011 = feature[feature_index(x0, y1, z1)];
//     auto f100 = feature[feature_index(x1, y0, z0)];
//     auto f101 = feature[feature_index(x1, y0, z1)];
//     auto f110 = feature[feature_index(x1, y1, z0)];
//     auto f111 = feature[feature_index(x1, y1, z1)];


//     // gradients
//     auto gg_xyz = *(float3*)(grad_grad_query + b * 3);
//     auto ggx = gg_xyz.x;
//     auto ggy = gg_xyz.y;
//     auto ggz = gg_xyz.z;
//     auto go = grad_output[n];
    
//     auto ti = go * sy * sz 
//       * (p0 * (f000 - f001 - f010 + f011) + p1 * (f100 - f101 - f110 + f111));
//     auto tj = go * sx * sz
//       * (q0 * (f000 - f001 - f100 + f101) + q1 * (f010 - f011 - f110 + f111));
//     auto tk = go * sx * sy
//       * (r0 * (f000 - f010 - f100 + f110) + r1 * (f001 - f011 - f101 + f111));

//     auto gx = ggy * tk + ggz * tj;
//     auto gy = ggz * ti + ggx * tk;
//     auto gz = ggx * tj + ggy * ti;

//     atomicAdd(grad_query + b * 3, gx);
//     atomicAdd(grad_query + b * 3 + 1, gy);
//     atomicAdd(grad_query + b * 3 + 2, gz);
//   }
// }


// void grad_query_grad_query(int N, int64_t grad_query_ptr, 
//                            int64_t grad_grad_query_ptr, 
//                            int64_t grad_output_ptr, 
//                            int64_t query_ptr, int64_t feature_ptr, 
//                            int G, int D, 
//                            std::vector<float> min, std::vector<float> max, 
//                            bool boundary_check, bool accum) {
//   auto grad_query_buff = reinterpret_cast<float*>(grad_query_ptr);
//   auto grad_grad_query_buff = reinterpret_cast<float*>(grad_grad_query_ptr);
//   auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
//   auto query_buff = reinterpret_cast<float*>(query_ptr);
//   auto feature_buff = reinterpret_cast<float*>(feature_ptr);

//   NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_query_grad_query, N, 
//                                  grad_query_buff,
//                                  grad_grad_query_buff,
//                                  grad_output_buff, 
//                                  query_buff, feature_buff, 
//                                  G, D, 
//                                  make_float3(min[0], min[1], min[2]),
//                                  make_float3(max[0], max[1], max[2]),
//                                  boundary_check);
// }
  
// 1-3. grad_query wrt feature
__global__
void kernel_grad_query_grad_feature(int N, float *grad_feature, 
                                    const float *grad_grad_query, 
                                    const float *grad_output, 
                                    const float *query, 
                                    int G, int D, 
                                    float3 min, float3 max, 
                                    bool boundary_check) {
  auto stride_u = G * D;
  auto stride_v = D;
  auto G1 = G - 1.f;
  auto grid_sizes1 = make_float3(G1, G1, G1);
  
  NBLA_CUDA_KERNEL_LOOP(n, N) {  // N = (B * R * N) * (D * 3)
    auto n_idx = flat_to_3d(n, D);
    auto b = n_idx.x, d = n_idx.y, i = n_idx.z;

    auto querys = *(float3*)(query + b * 3);
  
    // continuous point
    auto scales = grid_sizes1 / (max - min);
    auto xyz = (querys - min) * scales;

    // discrete points
    auto xyz0 = floorf(xyz);    
    xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
    xyz0 = fminf(xyz0, grid_sizes1);
    auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);

    // coefficients
    auto pqr0 = xyz1 - xyz;
    auto pqr1 = 1.f - pqr0;

    // gradients
    auto gg_xyz = *(float3*)(grad_grad_query + b * 3);
    auto gguv = select_gguv(i, gg_xyz);
    auto ggu = gguv.x, ggv = gguv.y;
    auto scale = select_scales(i, scales);
    auto su = scale.x, sv = scale.y;

    auto ograd = grad_output[n];
    auto grad_feature_i = grad_feature + i * (G * G * D);
    auto compute_grad = [&](const uint u, const uint v,
                            const float a, const float b) {
      auto f_idx = (u * stride_u) + (v * stride_v) + d;
      atomicAdd(grad_feature_i + f_idx, ograd * (ggu* su * a + ggv * sv * b));
    };

    auto locs = select_locations(i, xyz0, xyz1);
    auto coef = select_coefficients(i, pqr0, pqr1);
    auto u0 = locs.x, u1 = locs.y, v0 = locs.z, v1 = locs.w;
    auto a0 = coef.x, a1 = coef.y, b0 = coef.z, b1 = coef.w;

    compute_grad(u0, v0, -b0, -a0);
    compute_grad(u0, v1, -b1, +a0);
    compute_grad(u1, v0, +b0, -a1);
    compute_grad(u1, v1, +b1, +a1);
  }

}

void grad_query_grad_feature(int N, int64_t grad_feature_ptr, 
                             int64_t grad_grad_query_ptr, 
                             int64_t grad_output_ptr, 
                             int64_t query_ptr, 
                             int G, int D, 
                             std::vector<float> min, std::vector<float> max, 
                             bool boundary_check, bool accum) {
  auto grad_feature_buff = reinterpret_cast<float*>(grad_feature_ptr);
  auto grad_grad_query_buff = reinterpret_cast<float*>(grad_grad_query_ptr);
  auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_query_grad_feature, N, 
                                 grad_feature_buff,
                                 grad_grad_query_buff,
                                 grad_output_buff, 
                                 query_buff, 
                                 G, D, 
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}

// // 2-1. grad_feature wrt grad_output
// template<bool accum = false>
// __global__
// void kernel_grad_feature_grad_grad_output(int N, float *grad_grad_output, 
//                                           const float *grad_grad_feature, 
//                                           const float *query,
//                                           int G, int D, 
//                                           float3 min, float3 max, 
//                                           bool boundary_check) {
//   auto Gy0 = grid_sizes.y;
//   auto Gz0 = grid_sizes.z;
//   auto stride_x = Gy0 * Gz0 * D;
//   auto stride_y = Gz0 * D;
//   auto stride_z = D;
//   auto grid_sizes1 = to_float3(grid_sizes) - 1.f;
  
//   NBLA_CUDA_KERNEL_LOOP(n, N) {
//     auto b = n / D;
//     auto d = n - b * D;

//     auto querys = *(float3*)(query + b * 3);
  
//     // continuous point   
//     auto scales = grid_sizes1 / (max - min);
//     auto xyz = (querys - min) * scales;

//     // discrete points
//     auto xyz0 = floorf(xyz);    
//     xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
//     xyz0 = fminf(xyz0, grid_sizes1);
//     auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);

//     // coefficients
//     auto pqr0 = xyz1 - xyz;
//     auto pqr1 = 1.f - pqr0;

//     // scalars
//     uint x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;
//     uint x1 = xyz1.x, y1 = xyz1.y, z1 = xyz1.z;
//     auto p0 = pqr0.x, q0 = pqr0.y, r0 = pqr0.z;
//     auto p1 = pqr1.x, q1 = pqr1.y, r1 = pqr1.z;
    
//     // grid features
//     auto feature_index = [&](const uint x, const uint y, const uint z) {
//       return (x * stride_x) + (y * stride_y) + (z * stride_z + d);
//     };
    
//     auto ggf000 = grad_grad_feature[feature_index(x0, y0, z0)];
//     auto ggf001 = grad_grad_feature[feature_index(x0, y0, z1)];
//     auto ggf010 = grad_grad_feature[feature_index(x0, y1, z0)];
//     auto ggf011 = grad_grad_feature[feature_index(x0, y1, z1)];
//     auto ggf100 = grad_grad_feature[feature_index(x1, y0, z0)];
//     auto ggf101 = grad_grad_feature[feature_index(x1, y0, z1)];
//     auto ggf110 = grad_grad_feature[feature_index(x1, y1, z0)];
//     auto ggf111 = grad_grad_feature[feature_index(x1, y1, z1)];
    
//     // linear interpolation
//     auto ggo = p0 * q0 * r0 * ggf000
//       + p0 * q0 * r1 * ggf001
//       + p0 * q1 * r0 * ggf010
//       + p0 * q1 * r1 * ggf011
//       + p1 * q0 * r0 * ggf100
//       + p1 * q0 * r1 * ggf101
//       + p1 * q1 * r0 * ggf110
//       + p1 * q1 * r1 * ggf111;

//     grad_grad_output[n] = accum ? grad_grad_output[n] + ggo : ggo;
//   }
// }

// void grad_feature_grad_grad_output(int N, int64_t grad_grad_output_ptr, 
//                                    int64_t grad_grad_feature_ptr, 
//                                    int64_t query_ptr, 
//                                    int G, int D, 
//                                    std::vector<float> min, std::vector<float> max, 
//                                    bool boundary_check, bool accum) {
//   auto grad_grad_output_buff = reinterpret_cast<float*>(grad_grad_output_ptr);
//   auto grad_grad_feature_buff = reinterpret_cast<float*>(grad_grad_feature_ptr);
//   auto query_buff = reinterpret_cast<float*>(query_ptr);

//   auto kernel = accum 
//     ? kernel_grad_feature_grad_grad_output<true> 
//     : kernel_grad_feature_grad_grad_output<false>;
//   NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel, N, 
//                                  grad_grad_output_buff,
//                                  grad_grad_feature_buff,
//                                  query_buff, 
//                                  G, D, 
//                                  make_float3(min[0], min[1], min[2]),
//                                  make_float3(max[0], max[1], max[2]),
//                                  boundary_check);
// }

// // 2-2. grad_feature wrt query
// __global__
// void kernel_grad_feature_grad_query(int N, float *grad_query, 
//                                     const float *grad_grad_feature, 
//                                     const float *grad_output, 
//                                     const float *query,
//                                     int G, int D, 
//                                     float3 min, float3 max, 
//                                     bool boundary_check) {
//   auto Gy0 = grid_sizes.y;
//   auto Gz0 = grid_sizes.z;
//   auto stride_x = Gy0 * Gz0 * D;
//   auto stride_y = Gz0 * D;
//   auto stride_z = D;
//   auto grid_sizes1 = to_float3(grid_sizes) - 1.f;
  
//   NBLA_CUDA_KERNEL_LOOP(n, N) {
//     auto b = n / D;
//     auto d = n - b * D;

//     auto querys = *(float3*)(query + b * 3);
  
//     // continuous point   
//     auto scales = grid_sizes1 / (max - min);
//     auto xyz = (querys - min) * scales;

//     // discrete points
//     auto xyz0 = floorf(xyz);    
//     xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
//     xyz0 = fminf(xyz0, grid_sizes1);
//     auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);

//     // coefficients
//     auto pqr0 = xyz1 - xyz;
//     auto pqr1 = 1.f - pqr0;

//     // scalars
//     uint x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;
//     uint x1 = xyz1.x, y1 = xyz1.y, z1 = xyz1.z;
//     auto p0 = pqr0.x, q0 = pqr0.y, r0 = pqr0.z;
//     auto p1 = pqr1.x, q1 = pqr1.y, r1 = pqr1.z;
//     auto sx = scales.x, sy = scales.y, sz = scales.z;

//     // grid features
//     auto feature_index = [&](const uint x, const uint y, const uint z) {
//       return x * stride_x + y * stride_y + z * stride_z + d;
//     };
    
//     auto ggf000 = grad_grad_feature[feature_index(x0, y0, z0)];
//     auto ggf001 = grad_grad_feature[feature_index(x0, y0, z1)];
//     auto ggf010 = grad_grad_feature[feature_index(x0, y1, z0)];
//     auto ggf011 = grad_grad_feature[feature_index(x0, y1, z1)];
//     auto ggf100 = grad_grad_feature[feature_index(x1, y0, z0)];
//     auto ggf101 = grad_grad_feature[feature_index(x1, y0, z1)];
//     auto ggf110 = grad_grad_feature[feature_index(x1, y1, z0)];
//     auto ggf111 = grad_grad_feature[feature_index(x1, y1, z1)];

//     // gradients
//     auto ograd = grad_output[n];
//     auto compute_grad = [&](float scale, float a0, float a1, float b0, float b1, 
//                             float d00, float d01, float d10, float d11) {
//       return ograd * scale * (a0 * b0 * d00 + a0 * b1 * d01 + a1 * b0 * d10 + a1 * b1 * d11);
//     };
//     auto gx = compute_grad(sx, q0, q1, r0, r1, 
//                            (ggf100 - ggf000), 
//                            (ggf101 - ggf001), 
//                            (ggf110 - ggf010), 
//                            (ggf111 - ggf011));
//     auto gy = compute_grad(sy, p0, p1, r0, r1, 
//                            (ggf010 - ggf000), 
//                            (ggf011 - ggf001), 
//                            (ggf110 - ggf100), 
//                            (ggf111 - ggf101));
//     auto gz = compute_grad(sz, p0, p1, q0, q1, 
//                            (ggf001 - ggf000), 
//                            (ggf011 - ggf010), 
//                            (ggf101 - ggf100), 
//                            (ggf111 - ggf110));
//     atomicAdd(grad_query + b * 3, gx);
//     atomicAdd(grad_query + b * 3 + 1, gy);
//     atomicAdd(grad_query + b * 3 + 2, gz);
//   }  
// }


// void grad_feature_grad_query(int N, int64_t grad_query_ptr, 
//                              int64_t grad_grad_feature_ptr, 
//                              int64_t grad_output_ptr, 
//                              int64_t query_ptr, 
//                              int G, int D, 
//                              std::vector<float> min, std::vector<float> max, 
//                              bool boundary_check, bool accum) {
//   auto grad_query_buff = reinterpret_cast<float*>(grad_query_ptr);
//   auto grad_grad_feature_buff = reinterpret_cast<float*>(grad_grad_feature_ptr);
//   auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
//   auto query_buff = reinterpret_cast<float*>(query_ptr);

//   NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_feature_grad_query, N, 
//                                  grad_query_buff,
//                                  grad_grad_feature_buff,
//                                  grad_output_buff, 
//                                  query_buff, 
//                                  G, D, 
//                                  make_float3(min[0], min[1], min[2]),
//                                  make_float3(max[0], max[1], max[2]),
//                                  boundary_check);
// }

} // triplane_feauture

PYBIND11_MODULE(triplane_feature_cuda, m) {
  m.doc() = "Interpolation by query on grid";
  // forward
  m.def("query_on_triplane", &triplane_feature::query_on_triplane, "Interpolation by query on grid");

  // 1st-order gradient
  m.def("grad_query", &triplane_feature::grad_query, "");
  m.def("grad_feature", &triplane_feature::grad_feature, "");

  // // 2nd-order gradient of 1st-order gradient wrt query
  m.def("grad_query_grad_grad_output", 
        &triplane_feature::grad_query_grad_grad_output, "");
  // m.def("grad_query_grad_query", 
  //       &triplane_feature::grad_query_grad_query, "");
  m.def("grad_query_grad_feature", 
        &triplane_feature::grad_query_grad_feature, "");

  // 2nd-order gradient of 1st-order gradient wrt feature
  // m.def("grad_feature_grad_grad_output", 
  //       &triplane_feature::grad_feature_grad_grad_output, "");
  // m.def("grad_feature_grad_query", 
  //       &triplane_feature::grad_feature_grad_query, "");
}
