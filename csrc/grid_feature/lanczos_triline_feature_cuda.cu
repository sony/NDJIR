#include "hip/hip_runtime.h"
// Copyright 2022 Sony Group Corporation.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <pybind11/pybind11.h>

#include <pybind11/stl.h>
#include <hip/hip_vector_types.h>
#include <cuda_common.cuh>
#include <grid_feature/common.cuh>
#include <grid_feature/common_triline.cuh>

namespace py = pybind11;

using namespace grid_feature;
using namespace grid_feature::triline;

/***
    Forward
 ***/

namespace triline_feature {

//TODO: optimize access pattern like voxel hash

template<int window_size = 2>
__global__
void kernel_query_on_triline(int N, float *output, const float *query, const float *feature,
                                 int G, int D, 
                                 float3 min, float3 max, 
                                 bool boundary_check) {
                                 
  auto stride_u = D;
  auto G1 = G - 1.f;
  auto grid_sizes1 = make_float3(G1, G1, G1);
  
  NBLA_CUDA_KERNEL_LOOP(n, N) {  // N = (B * R * N) * (D * 3)
    auto n_idx = flat_to_3d(n, D);
    auto b = n_idx.x, d = n_idx.y, l = n_idx.z;

    auto querys = *(float3*)(query + b * 3);
  
    // continuous point
    auto scales = grid_sizes1 / (max - min);
    auto xyz = (querys - min) * scales;
    auto u = select_location(l, xyz);

    // discrete points
    auto xyz0 = floorf(xyz);
    auto u0 = select_location(l, xyz0);

    // grid features
    auto feature_index = [&](const uint u) {
      return (u * stride_u) + d;
    };

    auto feature_l = feature + l * (G * D);
    auto f = 0.f;
    auto w = window_size;

    for (auto i = -w + 1; i <= w; ++i) {
      auto ui = clamp(u0 + i, 0.f, G1);
      auto ci = lanczos(u - ui, w);

      auto fi = feature_l[feature_index(ui)];
      f += ci * fi;
    }
    output[n] = f;
  }
}


void query_on_triline(int N, int64_t output_ptr, int64_t query_ptr, int64_t feature_ptr, 
                          int G, int D, 
                          std::vector<float> min, std::vector<float> max, 
                          bool boundary_check) {
  auto output_buff = reinterpret_cast<float*>(output_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);
  auto feature_buff = reinterpret_cast<float*>(feature_ptr);

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_query_on_triline, N, 
                                 output_buff, query_buff, feature_buff, 
                                 G, D, 
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}


/***
    Backward (1st-order)
    1. wrt query
    2. wrt feature
 ***/

template<int window_size = 2>
__global__
void kernel_grad_query(int N, float *grad_query, 
                       const float *grad_output, 
                       const float *query, const float *feature,
                       int G, int D, 
                       float3 min, float3 max, 
                       bool boundary_check) {
  auto stride_u = D;
  auto G1 = G - 1.f;
  auto grid_sizes1 = make_float3(G1, G1, G1);

  NBLA_CUDA_KERNEL_LOOP(n, N) {  // N = (B * R * N) * (D * 3)
    auto n_idx = flat_to_3d(n, D);
    auto b = n_idx.x, d = n_idx.y, l = n_idx.z;

    auto querys = *(float3*)(query + b * 3);
  
    // continuous point
    auto scales = grid_sizes1 / (max - min);
    auto xyz = (querys - min) * scales;
    auto u = select_location(l, xyz);
    auto su = select_scale(l, scales);

    // discrete points
    auto xyz0 = floorf(xyz);
    auto u0 = select_location(l, xyz0);

    // grid features
    auto feature_index = [&](const uint u) {
      return (u * stride_u) + d;
    };

    auto feature_l = feature + l * (G * D);
    auto gu = 0.f;
    auto w = window_size;

    for (auto i = -w + 1; i <= w; ++i) {
      auto ui = clamp(u0 + i, 0.f, G1);
      auto ci = lanczos(u - ui, w);
      auto gci = grad_coefficient(u - ui, w);

      auto fi = feature_l[feature_index(ui)];
      gu += su * gci * fi;
    }
    auto ograd = grad_output[n];
    gu = ograd * gu;
    atomicAdd(grad_query + b * 3 + l, gu);
  }
}

void grad_query(int N, int64_t grad_query_ptr, 
                int64_t grad_output_ptr, int64_t query_ptr, int64_t feature_ptr, 
                int G, int D, 
                std::vector<float> min, std::vector<float> max, 
                bool boundary_check, bool accum) {
  auto grad_query_buff = reinterpret_cast<float*>(grad_query_ptr);
  auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);
  auto feature_buff = reinterpret_cast<float*>(feature_ptr);

  if (!accum) {
    auto size = N / D;
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_zero, size, grad_query_buff);
  }
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_query, N, 
                                 grad_query_buff, 
                                 grad_output_buff, query_buff, feature_buff, 
                                 G, D, 
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}


template<int window_size = 2>
__global__
void kernel_grad_feature(int N, float *grad_feature, 
                         const float *grad_output, 
                         const float *query, 
                         int G, int D, 
                         float3 min, float3 max, 
                         bool boundary_check) {
  auto stride_u = D;
  auto G1 = G - 1.f;
  auto grid_sizes1 = make_float3(G1, G1, G1);

  NBLA_CUDA_KERNEL_LOOP(n, N) {  // N = (B * R * N) * (D * 3)
    auto n_idx = flat_to_3d(n, D);
    auto b = n_idx.x, d = n_idx.y, l = n_idx.z;

    auto querys = *(float3*)(query + b * 3);
  
    // continuous point
    auto scales = grid_sizes1 / (max - min);
    auto xyz = (querys - min) * scales;
    auto u = select_location(l, xyz);

    // discrete points
    auto xyz0 = floorf(xyz);
    auto u0 = select_location(l, xyz0);

    // gradietns features
    auto ograd = grad_output[n];
    auto grad_feature_l = grad_feature + l * (G * D);
    auto compute_grad = [&](const uint u, const float a) {
      auto f_idx = (u * stride_u) + d;
      atomicAdd(grad_feature_l + f_idx, ograd * a);
    };

    auto w = window_size;

    for (auto i = -w + 1; i <= w; ++i) {
      auto ui = clamp(u0 + i, 0.f, G1);
      auto ci = lanczos(u - ui, w);
      compute_grad(ui, ci);
    }
  }
}


void grad_feature(int N, int64_t grad_feature_ptr, 
                  int64_t grad_output_ptr, int64_t query_ptr, 
                  int G, int D, 
                  std::vector<float> min, std::vector<float> max, 
                  bool boundary_check, bool accum) {
  auto grad_feature_buff = reinterpret_cast<float*>(grad_feature_ptr);
  auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);

  
  if (!accum) {
    auto size = 3 * G * D;
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_zero, size, grad_feature_buff);
  }

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_feature, N, 
                                 grad_feature_buff, 
                                 grad_output_buff, query_buff, 
                                 G, D, 
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}


/***
    Backward (2nd-order)
    1-1. grad_query wrt grad_output
    // 1-2. grad_query wrt query
    1-3. grad_query wrt feature
    // 2-1. grad_feature wrt grad_output
    // 2-2. grad_feature wrt query

    naming rule, kernel_<backward_function_name>_grad_<wrt_input>
    
 ***/


// 1-1. grad_query wrt grad_output
template<bool accum = false, int window_size = 2>
__global__
void kernel_grad_query_grad_grad_output(int N, float *grad_grad_output, 
                                        const float *grad_grad_query, 
                                        const float *query, const float *feature,
                                        int G, int D, 
                                        float3 min, float3 max, 
                                        bool boundary_check) {
  auto stride_u = D;
  auto G1 = G - 1.f;
  auto grid_sizes1 = make_float3(G1, G1, G1);

  NBLA_CUDA_KERNEL_LOOP(n, N) {  // N = (B * R * N) * (D * 3)
    auto n_idx = flat_to_3d(n, D);
    auto b = n_idx.x, d = n_idx.y, l = n_idx.z;

    auto querys = *(float3*)(query + b * 3);
  
    // continuous point
    auto scales = grid_sizes1 / (max - min);
    auto xyz = (querys - min) * scales;
    auto u = select_location(l, xyz);
    auto su = select_scale(l, scales);

    // discrete points
    auto xyz0 = floorf(xyz);
    auto u0 = select_location(l, xyz0);

    // grid features
    auto feature_index = [&](const uint u) {
      return (u * stride_u) + d;
    };

    auto feature_l = feature + l * (G * D);
    auto gu = 0.f;
    auto w = window_size;

    for (auto i = -w + 1; i <= w; ++i) {
      auto ui = clamp(u0 + i, 0.f, G1);
      auto ci = lanczos(u - ui, w);
      auto gci = grad_coefficient(u - ui, w);
      
      auto fi = feature_l[feature_index(ui)];
      gu += su * gci * fi;
    }

    auto ggq = *(float3*)(grad_grad_query + b * 3);
    auto ggu = select_ggu(l, ggq);
    auto ggo = ggu * gu;
    grad_grad_output[n] = accum ? grad_grad_output[n] + ggo : ggo;

  }
}


void grad_query_grad_grad_output(int N, int64_t grad_grad_output_ptr, 
                                 int64_t grad_grad_query_ptr, 
                                 int64_t query_ptr, int64_t feature_ptr, 
                                 int G, int D, 
                                 std::vector<float> min, std::vector<float> max, 
                                 bool boundary_check, bool accum) {
  auto grad_grad_output_buff = reinterpret_cast<float*>(grad_grad_output_ptr);
  auto grad_grad_query_buff = reinterpret_cast<float*>(grad_grad_query_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);
  auto feature_buff = reinterpret_cast<float*>(feature_ptr);

  auto kernel = accum 
    ? kernel_grad_query_grad_grad_output<true> 
    : kernel_grad_query_grad_grad_output<false>;
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel, N, 
                                 grad_grad_output_buff, 
                                 grad_grad_query_buff, 
                                 query_buff, feature_buff, 
                                 G, D, 
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}

// // 1-2. grad_query wrt query
// __global__
// void kernel_grad_query_grad_query(int N, float *grad_query, 
//                                   const float *grad_grad_query, 
//                                   const float *grad_output, 
//                                   const float *query, const float *feature,
//                                   int G, int D, 
//                                   float3 min, float3 max, 
//                                   bool boundary_check) {
//   auto Gy0 = grid_sizes.y;
//   auto Gz0 = grid_sizes.z;
//   auto stride_x = Gy0 * Gz0 * D;
//   auto stride_y = Gz0 * D;
//   auto stride_z = D;
//   auto grid_sizes1 = to_float3(grid_sizes) - 1.f;
  
//   NBLA_CUDA_KERNEL_LOOP(n, N) {

//     auto b = n / D;
//     auto d = n - b * D;

//     auto querys = *(float3*)(query + b * 3);
  
//     // continuous point   
//     auto scales = grid_sizes1 / (max - min);
//     auto xyz = (querys - min) * scales;

//     // discrete points
//     auto xyz0 = floorf(xyz);    
//     xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
//     xyz0 = fminf(xyz0, grid_sizes1);
//     auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);

//     // coefficients
//     auto pqr0 = 0.5 * cos(M_PI * (xyz - xyz0)) + 0.5;
//     auto pqr1 = 1.f - pqr0;

//     // scalars
//     uint x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;
//     uint x1 = xyz1.x, y1 = xyz1.y, z1 = xyz1.z;
//     auto p0 = pqr0.x, q0 = pqr0.y, r0 = pqr0.z;
//     auto p1 = pqr1.x, q1 = pqr1.y, r1 = pqr1.z;
//     auto sx = scales.x, sy = scales.y, sz = scales.z;

//     // grid features
//     auto feature_index = [&](const uint x, const uint y, const uint z) {
//       return x * stride_x + y * stride_y + z * stride_z + d;
//     };
    
//     auto f000 = feature[feature_index(x0, y0, z0)];
//     auto f001 = feature[feature_index(x0, y0, z1)];
//     auto f010 = feature[feature_index(x0, y1, z0)];
//     auto f011 = feature[feature_index(x0, y1, z1)];
//     auto f100 = feature[feature_index(x1, y0, z0)];
//     auto f101 = feature[feature_index(x1, y0, z1)];
//     auto f110 = feature[feature_index(x1, y1, z0)];
//     auto f111 = feature[feature_index(x1, y1, z1)];


//     // gradients
//     auto gg_xyz = *(float3*)(grad_grad_query + b * 3);
//     auto ggx = gg_xyz.x;
//     auto ggy = gg_xyz.y;
//     auto ggz = gg_xyz.z;
//     auto go = grad_output[n];
    
//     auto ti = go * sy * sz 
//       * (p0 * (f000 - f001 - f010 + f011) + p1 * (f100 - f101 - f110 + f111));
//     auto tj = go * sx * sz
//       * (q0 * (f000 - f001 - f100 + f101) + q1 * (f010 - f011 - f110 + f111));
//     auto tk = go * sx * sy
//       * (r0 * (f000 - f010 - f100 + f110) + r1 * (f001 - f011 - f101 + f111));

//     auto gx = ggy * tk + ggz * tj;
//     auto gy = ggz * ti + ggx * tk;
//     auto gz = ggx * tj + ggy * ti;

//     atomicAdd(grad_query + b * 3, gx);
//     atomicAdd(grad_query + b * 3 + 1, gy);
//     atomicAdd(grad_query + b * 3 + 2, gz);
//   }
// }


// void grad_query_grad_query(int N, int64_t grad_query_ptr, 
//                            int64_t grad_grad_query_ptr, 
//                            int64_t grad_output_ptr, 
//                            int64_t query_ptr, int64_t feature_ptr, 
//                            int G, int D, 
//                            std::vector<float> min, std::vector<float> max, 
//                            bool boundary_check, bool accum) {
//   auto grad_query_buff = reinterpret_cast<float*>(grad_query_ptr);
//   auto grad_grad_query_buff = reinterpret_cast<float*>(grad_grad_query_ptr);
//   auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
//   auto query_buff = reinterpret_cast<float*>(query_ptr);
//   auto feature_buff = reinterpret_cast<float*>(feature_ptr);

//   NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_query_grad_query, N, 
//                                  grad_query_buff,
//                                  grad_grad_query_buff,
//                                  grad_output_buff, 
//                                  query_buff, feature_buff, 
//                                  G, D, 
//                                  make_float3(min[0], min[1], min[2]),
//                                  make_float3(max[0], max[1], max[2]),
//                                  boundary_check);
// }
  
// 1-3. grad_query wrt feature
template<int window_size = 2>
__global__
void kernel_grad_query_grad_feature(int N, float *grad_feature, 
                                    const float *grad_grad_query, 
                                    const float *grad_output, 
                                    const float *query, 
                                    int G, int D, 
                                    float3 min, float3 max, 
                                    bool boundary_check) {
  auto stride_u = D;
  auto G1 = G - 1.f;
  auto grid_sizes1 = make_float3(G1, G1, G1);
  
  NBLA_CUDA_KERNEL_LOOP(n, N) {  // N = (B * R * N) * (D * 3)
    auto n_idx = flat_to_3d(n, D);
    auto b = n_idx.x, d = n_idx.y, l = n_idx.z;

    auto querys = *(float3*)(query + b * 3);
  
    // continuous point
    auto scales = grid_sizes1 / (max - min);
    auto xyz = (querys - min) * scales;
    auto u = select_location(l, xyz);
    auto su = select_scale(l, scales);

    // discrete points
    auto xyz0 = floorf(xyz);
    auto u0 = select_location(l, xyz0);

    // gradients
    auto ggq = *(float3*)(grad_grad_query + b * 3);
    auto ggu = select_ggu(l, ggq);
    auto ograd = grad_output[n];
    auto grad_feature_l = grad_feature + l * (G * D);
    auto compute_grad = [&](const uint u, const float a) {
      auto f_idx = (u * stride_u) + d;
      atomicAdd(grad_feature_l + f_idx, ograd * ggu * su * a);
    };
    
    auto w = window_size;

    for (auto i = -w + 1; i <= w; ++i) {
      auto ui = clamp(u0 + i, 0.f, G1);
      auto ci = lanczos(u - ui, w);
      auto gci = grad_coefficient(u - ui, w);

      compute_grad(ui, gci);
    }
  }

}

void grad_query_grad_feature(int N, int64_t grad_feature_ptr, 
                             int64_t grad_grad_query_ptr, 
                             int64_t grad_output_ptr, 
                             int64_t query_ptr, 
                             int G, int D, 
                             std::vector<float> min, std::vector<float> max, 
                             bool boundary_check, bool accum) {
  auto grad_feature_buff = reinterpret_cast<float*>(grad_feature_ptr);
  auto grad_grad_query_buff = reinterpret_cast<float*>(grad_grad_query_ptr);
  auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_query_grad_feature, N, 
                                 grad_feature_buff,
                                 grad_grad_query_buff,
                                 grad_output_buff, 
                                 query_buff, 
                                 G, D, 
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}

// // 2-1. grad_feature wrt grad_output
// template<bool accum = false>
// __global__
// void kernel_grad_feature_grad_grad_output(int N, float *grad_grad_output, 
//                                           const float *grad_grad_feature, 
//                                           const float *query,
//                                           int G, int D, 
//                                           float3 min, float3 max, 
//                                           bool boundary_check) {
//   auto Gy0 = grid_sizes.y;
//   auto Gz0 = grid_sizes.z;
//   auto stride_x = Gy0 * Gz0 * D;
//   auto stride_y = Gz0 * D;
//   auto stride_z = D;
//   auto grid_sizes1 = to_float3(grid_sizes) - 1.f;
  
//   NBLA_CUDA_KERNEL_LOOP(n, N) {
//     auto b = n / D;
//     auto d = n - b * D;

//     auto querys = *(float3*)(query + b * 3);
  
//     // continuous point   
//     auto scales = grid_sizes1 / (max - min);
//     auto xyz = (querys - min) * scales;

//     // discrete points
//     auto xyz0 = floorf(xyz);    
//     xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
//     xyz0 = fminf(xyz0, grid_sizes1);
//     auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);

//     // coefficients
//     auto pqr0 = 0.5 * cos(M_PI * (xyz - xyz0)) + 0.5;
//     auto pqr1 = 1.f - pqr0;

//     // scalars
//     uint x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;
//     uint x1 = xyz1.x, y1 = xyz1.y, z1 = xyz1.z;
//     auto p0 = pqr0.x, q0 = pqr0.y, r0 = pqr0.z;
//     auto p1 = pqr1.x, q1 = pqr1.y, r1 = pqr1.z;
    
//     // grid features
//     auto feature_index = [&](const uint x, const uint y, const uint z) {
//       return (x * stride_x) + (y * stride_y) + (z * stride_z + d);
//     };
    
//     auto ggf000 = grad_grad_feature[feature_index(x0, y0, z0)];
//     auto ggf001 = grad_grad_feature[feature_index(x0, y0, z1)];
//     auto ggf010 = grad_grad_feature[feature_index(x0, y1, z0)];
//     auto ggf011 = grad_grad_feature[feature_index(x0, y1, z1)];
//     auto ggf100 = grad_grad_feature[feature_index(x1, y0, z0)];
//     auto ggf101 = grad_grad_feature[feature_index(x1, y0, z1)];
//     auto ggf110 = grad_grad_feature[feature_index(x1, y1, z0)];
//     auto ggf111 = grad_grad_feature[feature_index(x1, y1, z1)];
    
//     // linear interpolation
//     auto ggo = p0 * q0 * r0 * ggf000
//       + p0 * q0 * r1 * ggf001
//       + p0 * q1 * r0 * ggf010
//       + p0 * q1 * r1 * ggf011
//       + p1 * q0 * r0 * ggf100
//       + p1 * q0 * r1 * ggf101
//       + p1 * q1 * r0 * ggf110
//       + p1 * q1 * r1 * ggf111;

//     grad_grad_output[n] = accum ? grad_grad_output[n] + ggo : ggo;
//   }
// }

// void grad_feature_grad_grad_output(int N, int64_t grad_grad_output_ptr, 
//                                    int64_t grad_grad_feature_ptr, 
//                                    int64_t query_ptr, 
//                                    int G, int D, 
//                                    std::vector<float> min, std::vector<float> max, 
//                                    bool boundary_check, bool accum) {
//   auto grad_grad_output_buff = reinterpret_cast<float*>(grad_grad_output_ptr);
//   auto grad_grad_feature_buff = reinterpret_cast<float*>(grad_grad_feature_ptr);
//   auto query_buff = reinterpret_cast<float*>(query_ptr);

//   auto kernel = accum 
//     ? kernel_grad_feature_grad_grad_output<true> 
//     : kernel_grad_feature_grad_grad_output<false>;
//   NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel, N, 
//                                  grad_grad_output_buff,
//                                  grad_grad_feature_buff,
//                                  query_buff, 
//                                  G, D, 
//                                  make_float3(min[0], min[1], min[2]),
//                                  make_float3(max[0], max[1], max[2]),
//                                  boundary_check);
// }

// // 2-2. grad_feature wrt query
// __global__
// void kernel_grad_feature_grad_query(int N, float *grad_query, 
//                                     const float *grad_grad_feature, 
//                                     const float *grad_output, 
//                                     const float *query,
//                                     int G, int D, 
//                                     float3 min, float3 max, 
//                                     bool boundary_check) {
//   auto Gy0 = grid_sizes.y;
//   auto Gz0 = grid_sizes.z;
//   auto stride_x = Gy0 * Gz0 * D;
//   auto stride_y = Gz0 * D;
//   auto stride_z = D;
//   auto grid_sizes1 = to_float3(grid_sizes) - 1.f;
  
//   NBLA_CUDA_KERNEL_LOOP(n, N) {
//     auto b = n / D;
//     auto d = n - b * D;

//     auto querys = *(float3*)(query + b * 3);
  
//     // continuous point   
//     auto scales = grid_sizes1 / (max - min);
//     auto xyz = (querys - min) * scales;

//     // discrete points
//     auto xyz0 = floorf(xyz);    
//     xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
//     xyz0 = fminf(xyz0, grid_sizes1);
//     auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);

//     // coefficients
//     auto pqr0 = 0.5 * cos(M_PI * (xyz - xyz0)) + 0.5;
//     auto pqr1 = 1.f - pqr0;

//     // scalars
//     uint x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;
//     uint x1 = xyz1.x, y1 = xyz1.y, z1 = xyz1.z;
//     auto p0 = pqr0.x, q0 = pqr0.y, r0 = pqr0.z;
//     auto p1 = pqr1.x, q1 = pqr1.y, r1 = pqr1.z;
//     auto sx = scales.x, sy = scales.y, sz = scales.z;

//     // grid features
//     auto feature_index = [&](const uint x, const uint y, const uint z) {
//       return x * stride_x + y * stride_y + z * stride_z + d;
//     };
    
//     auto ggf000 = grad_grad_feature[feature_index(x0, y0, z0)];
//     auto ggf001 = grad_grad_feature[feature_index(x0, y0, z1)];
//     auto ggf010 = grad_grad_feature[feature_index(x0, y1, z0)];
//     auto ggf011 = grad_grad_feature[feature_index(x0, y1, z1)];
//     auto ggf100 = grad_grad_feature[feature_index(x1, y0, z0)];
//     auto ggf101 = grad_grad_feature[feature_index(x1, y0, z1)];
//     auto ggf110 = grad_grad_feature[feature_index(x1, y1, z0)];
//     auto ggf111 = grad_grad_feature[feature_index(x1, y1, z1)];

//     // gradients
//     auto ograd = grad_output[n];
//     auto compute_grad = [&](float scale, float a0, float a1, float b0, float b1, 
//                             float d00, float d01, float d10, float d11) {
//       return ograd * scale * (a0 * b0 * d00 + a0 * b1 * d01 + a1 * b0 * d10 + a1 * b1 * d11);
//     };
//     auto gx = compute_grad(sx, q0, q1, r0, r1, 
//                            (ggf100 - ggf000), 
//                            (ggf101 - ggf001), 
//                            (ggf110 - ggf010), 
//                            (ggf111 - ggf011));
//     auto gy = compute_grad(sy, p0, p1, r0, r1, 
//                            (ggf010 - ggf000), 
//                            (ggf011 - ggf001), 
//                            (ggf110 - ggf100), 
//                            (ggf111 - ggf101));
//     auto gz = compute_grad(sz, p0, p1, q0, q1, 
//                            (ggf001 - ggf000), 
//                            (ggf011 - ggf010), 
//                            (ggf101 - ggf100), 
//                            (ggf111 - ggf110));
//     atomicAdd(grad_query + b * 3, gx);
//     atomicAdd(grad_query + b * 3 + 1, gy);
//     atomicAdd(grad_query + b * 3 + 2, gz);
//   }  
// }


// void grad_feature_grad_query(int N, int64_t grad_query_ptr, 
//                              int64_t grad_grad_feature_ptr, 
//                              int64_t grad_output_ptr, 
//                              int64_t query_ptr, 
//                              int G, int D, 
//                              std::vector<float> min, std::vector<float> max, 
//                              bool boundary_check, bool accum) {
//   auto grad_query_buff = reinterpret_cast<float*>(grad_query_ptr);
//   auto grad_grad_feature_buff = reinterpret_cast<float*>(grad_grad_feature_ptr);
//   auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
//   auto query_buff = reinterpret_cast<float*>(query_ptr);

//   NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_feature_grad_query, N, 
//                                  grad_query_buff,
//                                  grad_grad_feature_buff,
//                                  grad_output_buff, 
//                                  query_buff, 
//                                  G, D, 
//                                  make_float3(min[0], min[1], min[2]),
//                                  make_float3(max[0], max[1], max[2]),
//                                  boundary_check);
// }

} // triline_feauture

PYBIND11_MODULE(lanczos_triline_feature_cuda, m) {
  m.doc() = "Interpolation by query on grid";
  // forward
  m.def("query_on_triline", &triline_feature::query_on_triline, "Interpolation by query on grid");

  // 1st-order gradient
  m.def("grad_query", &triline_feature::grad_query, "");
  m.def("grad_feature", &triline_feature::grad_feature, "");

  // // 2nd-order gradient of 1st-order gradient wrt query
  m.def("grad_query_grad_grad_output", 
        &triline_feature::grad_query_grad_grad_output, "");
  // m.def("grad_query_grad_query", 
  //       &triline_feature::grad_query_grad_query, "");
  m.def("grad_query_grad_feature", 
        &triline_feature::grad_query_grad_feature, "");

  // 2nd-order gradient of 1st-order gradient wrt feature
  // m.def("grad_feature_grad_grad_output", 
  //       &triline_feature::grad_feature_grad_grad_output, "");
  // m.def("grad_feature_grad_query", 
  //       &triline_feature::grad_feature_grad_query, "");
}
