#include "hip/hip_runtime.h"
// Copyright 2022 Sony Group Corporation.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <pybind11/pybind11.h>

#include <pybind11/stl.h>
#include <cuda_common.cuh>
#include <hip/hip_vector_types.h>
#include <grid_feature/common.cuh>
#include <grid_feature/common_voxel_hash.cuh>

namespace py = pybind11;

using namespace grid_feature;
using namespace grid_feature::voxel_hash;

/***
    Forward
 ***/


namespace voxel_hash_feature {

// borrowed from
// https://github.com/NVlabs/tiny-cuda-nn/blob/4e0905311f1245c60b9fcc3a1c271ebb0ba50218/include/tiny-cuda-nn/encodings/grid.h#L77
__device__ inline
uint hash(uint x, uint y, uint z, int T) {
  constexpr uint32_t primes[3] = {1u, 2654435761u, 805459861u};  
	uint32_t result = 0;
	
  result ^= x * primes[0];
  result ^= y * primes[1];
  result ^= z * primes[2];

  auto index = result % T;
	return index;
}

/***
 hash index
 ***/

__global__
void kernel_hash_index(int N, float *output, const float *query, 
                       int T, 
                       bool boundary_check) {

  NBLA_CUDA_KERNEL_LOOP(n, N) { // N = B
    auto b = n;
    auto querys = *(float3*)(query + b * 3);
    uint x0 = querys.x, y0 = querys.y, z0 = querys.z;
    auto idx = hash(x0, y0, z0, T);
    *(output + b) = idx;
  }
}


void hash_index(int N, int64_t output_ptr, int64_t query_ptr, 
                int T, 
                bool boundary_check) {
  auto output_buff = reinterpret_cast<float*>(output_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_hash_index, N, 
                                 output_buff, query_buff, 
                                 T, 
                                 boundary_check);
}


__device__ inline
float3 cos(float3 data) {
  return make_float3(cosf(data.x), cosf(data.y), cosf(data.z));
}

__device__ inline
float3 sin(float3 data) {
  return make_float3(sinf(data.x), sinf(data.y), sinf(data.z));
}

__device__ inline
float sinc(float x) {
  if (x == 0.f)
    return 1.0;
  return sinf(x) / x;
}


__device__ inline
float lanczos(float x, int a) {
  auto z = M_PI * x;
  auto u = sinc(z);
  auto v = sinc(z / a);
  auto y = u * v;
  return y;
}

__device__ inline
float grad_coefficient(float x, int a) {
  if (x == 0.f)
    return 0.0;

  auto z0 = M_PI * x;
  auto z1 = M_PI * x / a;
  auto sinc_z0 = sinc(z0);
  auto sinc_z1 = sinc(z1);

  auto t0 = (cosf(z0) - sinc_z0) * sinc_z1;
  auto t1 = (cosf(z1) - sinc_z1) * sinc_z0;

  auto gc = (t0 + t1) / x;
  return gc;
}

/***
 forward
 ***/

template<int window_size = 2>
__global__
void kernel_voxel_hash_feature(int N, float *output, const float *query, const float *feature,
                               int G0, float growth_factor, int T0, int L, int D, 
                               float3 min, float3 max, 
                               bool boundary_check) {

  NBLA_CUDA_KERNEL_LOOP(n, N) { // N = L * (B * R * N)
    // query
    auto B = N / L;
    auto lb = flat_to_2d(n, B);
    auto l = lb.x, b = lb.y;
    auto querys = *(float3*)(query + b * 3);

    // variable feature hash table (T_l, D)
    auto G = compute_grid_size(G0, growth_factor, l);
    auto T = compute_table_size(G, T0);
    auto grid_sizes1 = make_float3(G, G, G) - 1.f;
    auto gx1 = grid_sizes1.x, gy1 = grid_sizes1.y, gz1 = grid_sizes1.z;

    // continuous point
    auto scales = grid_sizes1 / (max - min);
    auto xyz = (querys - min) * scales;
    auto x = xyz.x, y = xyz.y, z = xyz.z;

    // discrete points
    auto xyz0 = floorf(xyz);
    auto x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;

    // grid features
    auto feature_index = [&](const uint x, const uint y, const uint z, const uint d) {
      auto hindex = hash(x, y, z, T);
      return hindex * D + d;
    };

    auto num_params = compute_num_params(G0, growth_factor, T0, l, D);
    auto feature_l = feature + num_params;

    for (int d = 0; d < D; ++d) {
      auto f = 0.f;
      auto w = window_size;

      for (auto i = -w + 1; i <= w; ++i) {
        auto xi = clamp(x0 + i, 0.f, gx1);
        auto ci = lanczos(x - xi, w);

        for (auto j = -w + 1; j <= w; ++j) {
          auto yj = clamp(y0 + j, 0.f, gy1);
          auto cj = lanczos(y - yj, w);

          for (auto k = -w + 1; k <= w; ++k) {
            auto zk = clamp(z0 + k, 0.f, gz1);
            auto ck = lanczos(z - zk, w);

            auto f_ijk = feature_l[feature_index(xi, yj, zk, d)];
            auto c_ijk = ci * cj * ck;
            f += c_ijk * f_ijk;
          }
        }
      }
      
      auto o_idx = d * (L * B) + l * (B) + b;
      output[o_idx] = f;
    }
  }
}


void voxel_hash_feature(int N, int64_t output_ptr, int64_t query_ptr, int64_t feature_ptr, 
                        int G0, float growth_factor, int T0, int L, int D, 
                        std::vector<float> min, std::vector<float> max, 
                        bool boundary_check) {
  auto output_buff = reinterpret_cast<float*>(output_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);
  auto feature_buff = reinterpret_cast<float*>(feature_ptr);

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_voxel_hash_feature, N, 
                                 output_buff, query_buff, feature_buff, 
                                 G0, growth_factor, T0, L, D,
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}


/***
    Backward (1st-order)
    1. wrt query
    2. wrt feature
 ***/

template<int window_size = 2>
__global__
void kernel_grad_query(int N, float *grad_query, 
                       const float *grad_output, 
                       const float *query, const float *feature,
                       int G0, float growth_factor, int T0, int L, int D, 
                       float3 min, float3 max, 
                       bool boundary_check) {
  
  NBLA_CUDA_KERNEL_LOOP(n, N) { // N = L * (B * R * N)
    // query
    auto B = N / L;
    auto lb = flat_to_2d(n, B);
    auto l = lb.x, b = lb.y;
    auto querys = *(float3*)(query + b * 3);

    // variable feature hash table (T_l, D)
    auto G = compute_grid_size(G0, growth_factor, l);
    auto T = compute_table_size(G, T0);
    auto grid_sizes1 = make_float3(G, G, G) - 1.f;
    auto gx1 = grid_sizes1.x, gy1 = grid_sizes1.y, gz1 = grid_sizes1.z;

    // continuous point
    auto scales = grid_sizes1 / (max - min);
    auto sx = scales.x, sy = scales.y, sz = scales.z;
    auto xyz = (querys - min) * scales;
    auto x = xyz.x, y = xyz.y, z = xyz.z;

    // discrete points
    auto xyz0 = floorf(xyz);
    auto x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;

    // grid features
    auto feature_index = [&](const uint x, const uint y, const uint z, const uint d) {
      auto hindex = hash(x, y, z, T);
      return hindex * D + d;
    };
    
    auto num_params = compute_num_params(G0, growth_factor, T0, l, D);
    auto feature_l = feature + num_params;

    for (int d = 0; d < D; ++d) {
      auto gx = 0.f, gy = 0.f, gz = 0.f;
      auto w = window_size;

      for (auto i = -w + 1; i <= w; ++i) {
        auto xi = clamp(x0 + i, 0.f, gx1);
        auto ci = lanczos(x - xi, w);
        auto gci = grad_coefficient(x - xi, w);

        for (auto j = -w + 1; j <= w; ++j) {
          auto yj = clamp(y0 + j, 0.f, gy1);
          auto cj = lanczos(y - yj, w);
          auto gcj = grad_coefficient(y - yj, w);

          for (auto k = -w + 1; k <= w; ++k) {
            auto zk = clamp(z0 + k, 0.f, gz1);
            auto ck = lanczos(z - zk, w);
            auto gck = grad_coefficient(z - zk, w);

            auto f_ijk = feature_l[feature_index(xi, yj, zk, d)];

            gx += sx * gci * cj * ck * f_ijk;
            gy += sy * ci * gcj * ck * f_ijk;
            gz += sz * ci * cj * gck * f_ijk;
          }
        }
      }
      
      auto o_idx = d * (L * B) + l * (B) + b;
      auto ograd = grad_output[o_idx];
      gx = ograd * gx;
      gy = ograd * gy;
      gz = ograd * gz;
      atomicAdd(grad_query + b * 3, gx);
      atomicAdd(grad_query + b * 3 + 1, gy);
      atomicAdd(grad_query + b * 3 + 2, gz);
    }
  }
}

void grad_query(int N, int64_t grad_query_ptr, 
                int64_t grad_output_ptr, int64_t query_ptr, int64_t feature_ptr, 
                int G0, float growth_factor, int T0, int L, int D, 
                std::vector<float> min, std::vector<float> max, 
                bool boundary_check, bool accum) {
  auto grad_query_buff = reinterpret_cast<float*>(grad_query_ptr);
  auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);
  auto feature_buff = reinterpret_cast<float*>(feature_ptr);

  if (!accum) {
    auto size = N / L * 3;
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_zero, size, grad_query_buff);
  }
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_query, N, 
                                 grad_query_buff, 
                                 grad_output_buff, query_buff, feature_buff, 
                                 G0, growth_factor, T0, L, D,
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}


template<int window_size = 2>
__global__
void kernel_grad_feature(int N, float *grad_feature, 
                         const float *grad_output, 
                         const float *query, 
                         int G0, float growth_factor, int T0, int L, int D, 
                         float3 min, float3 max, 
                         bool boundary_check) {

  NBLA_CUDA_KERNEL_LOOP(n, N) { // N = L * (B * R * N)
    // query
    auto B = N / L;
    auto lb = flat_to_2d(n, B);
    auto l = lb.x, b = lb.y;
    auto querys = *(float3*)(query + b * 3);

    // variable feature hash table (T_l, D)
    auto G = compute_grid_size(G0, growth_factor, l);
    auto T = compute_table_size(G, T0);
    auto grid_sizes1 = make_float3(G, G, G) - 1.f;
    auto gx1 = grid_sizes1.x, gy1 = grid_sizes1.y, gz1 = grid_sizes1.z;

    // continuous point
    auto scales = grid_sizes1 / (max - min);
    auto xyz = (querys - min) * scales;
    auto x = xyz.x, y = xyz.y, z = xyz.z;

    // discrete points
    auto xyz0 = floorf(xyz);
    auto x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;

    // gradients
    auto feature_index = [&](const uint x, const uint y, const uint z, const uint d) {
      auto hindex = hash(x, y, z, T);
      return hindex * D + d;
    };

    auto num_params = compute_num_params(G0, growth_factor, T0, l, D);
    auto grad_feature_l = grad_feature + num_params;

    auto compute_grad = [&](const float ograd, const uint x, const uint y, const uint z, const uint d, 
                            const float p, const float q, const float r) {
      auto f_idx = feature_index(x, y, z, d);
      atomicAdd(grad_feature_l + f_idx, ograd * p * q * r);
    };

    for (int d = 0; d < D; ++d) {
      auto w = window_size;
      
      auto o_idx = d * (L * B) + l * (B) + b;
      auto ograd = grad_output[o_idx];

      for (auto i = -w + 1; i <= w; ++i) {
        auto xi = clamp(x0 + i, 0.f, gx1);
        auto ci = lanczos(x - xi, w);

        for (auto j = -w + 1; j <= w; ++j) {
          auto yj = clamp(y0 + j, 0.f, gy1);
          auto cj = lanczos(y - yj, w);

          for (auto k = -w + 1; k <= w; ++k) {
            auto zk = clamp(z0 + k, 0.f, gz1);
            auto ck = lanczos(z - zk, w);
            compute_grad(ograd, xi, yj, zk, d, ci, cj, ck);
          }
        }
      }
    }
  }
}


void grad_feature(int N, int64_t grad_feature_ptr, 
                  int64_t grad_output_ptr, int64_t query_ptr, 
                  int G0, float growth_factor, int T0, int L, int D, 
                  std::vector<float> min, std::vector<float> max, 
                  bool boundary_check, bool accum) {
  auto grad_feature_buff = reinterpret_cast<float*>(grad_feature_ptr);
  auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);
  
  if (!accum) {
    auto size = compute_num_params(G0, growth_factor, T0, L, D);
    NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_zero, size, grad_feature_buff);
  }

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_feature, N, 
                                 grad_feature_buff, 
                                 grad_output_buff, query_buff, 
                                 G0, growth_factor, T0, L, D,
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}


/***
    Backward (2nd-order)
    1-1. grad_query wrt grad_output
    // 1-2. grad_query wrt query
    1-3. grad_query wrt feature
    // 2-1. grad_feature wrt grad_output
    // 2-2. grad_feature wrt query

    naming rule, kernel_<backward_function_name>_grad_<wrt_input>
    
 ***/


// 1-1. grad_query wrt grad_output
template<bool accum = false, int window_size = 2>
__global__
void kernel_grad_query_grad_grad_output(int N, float *grad_grad_output, 
                                        const float *grad_grad_query, 
                                        const float *query, const float *feature,
                                        int G0, float growth_factor, int T0, int L, int D, 
                                        float3 min, float3 max, 
                                        bool boundary_check) {

  NBLA_CUDA_KERNEL_LOOP(n, N) { // N = L * (B * R * N)
    // query
    auto B = N / L;
    auto lb = flat_to_2d(n, B);
    auto l = lb.x, b = lb.y;
    auto querys = *(float3*)(query + b * 3);

    // variable feature hash table (T_l, D)
    auto G = compute_grid_size(G0, growth_factor, l);
    auto T = compute_table_size(G, T0);
    auto grid_sizes1 = make_float3(G, G, G) - 1.f;
    auto gx1 = grid_sizes1.x, gy1 = grid_sizes1.y, gz1 = grid_sizes1.z;

    // continuous point
    auto scales = grid_sizes1 / (max - min);
    auto sx = scales.x, sy = scales.y, sz = scales.z;
    auto xyz = (querys - min) * scales;
    auto x = xyz.x, y = xyz.y, z = xyz.z;

    // discrete points
    auto xyz0 = floorf(xyz);
    auto x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;

    // grid features
    auto feature_index = [&](const uint x, const uint y, const uint z, const uint d) {
      auto hindex = hash(x, y, z, T);
      return hindex * D + d;
    };

    auto num_params = compute_num_params(G0, growth_factor, T0, l, D);
    auto feature_l = feature + num_params;

    auto ggq = *(float3*)(grad_grad_query + b * 3);

    for (int d = 0; d < D; ++d) {

      auto gx = 0.f, gy = 0.f, gz = 0.f;
      auto w = window_size;

      for (auto i = -w + 1; i <= w; ++i) {
        auto xi = clamp(x0 + i, 0.f, gx1);
        auto ci = lanczos(x - xi, w);
        auto gci = grad_coefficient(x - xi, w);

        for (auto j = -w + 1; j <= w; ++j) {
          auto yj = clamp(y0 + j, 0.f, gy1);
          auto cj = lanczos(y - yj, w);
          auto gcj = grad_coefficient(y - yj, w);

          for (auto k = -w + 1; k <= w; ++k) {
            auto zk = clamp(z0 + k, 0.f, gz1);
            auto ck = lanczos(z - zk, w);
            auto gck = grad_coefficient(z - zk, w);

            auto f_ijk = feature_l[feature_index(xi, yj, zk, d)];

            gx += sx * gci * cj * ck * f_ijk;
            gy += sy * ci * gcj * ck * f_ijk;
            gz += sz * ci * cj * gck * f_ijk;
          }
        }
      }
      auto o_idx = d * (L * B) + l * (B) + b;
      auto ggo = ggq.x * gx + ggq.y * gy + ggq.z * gz;
      grad_grad_output[o_idx] = accum ? grad_grad_output[o_idx] + ggo : ggo;
    }
  }
}


void grad_query_grad_grad_output(int N, int64_t grad_grad_output_ptr, 
                                 int64_t grad_grad_query_ptr, 
                                 int64_t query_ptr, int64_t feature_ptr, 
                                 int G0, float growth_factor, int T0, int L, int D, 
                                 std::vector<float> min, std::vector<float> max, 
                                 bool boundary_check, bool accum) {
  auto grad_grad_output_buff = reinterpret_cast<float*>(grad_grad_output_ptr);
  auto grad_grad_query_buff = reinterpret_cast<float*>(grad_grad_query_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);
  auto feature_buff = reinterpret_cast<float*>(feature_ptr);

  auto kernel = accum 
    ? kernel_grad_query_grad_grad_output<true> 
    : kernel_grad_query_grad_grad_output<false>;
  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel, N, 
                                 grad_grad_output_buff, 
                                 grad_grad_query_buff, 
                                 query_buff, feature_buff, 
                                 G0, growth_factor, T0, L, D,
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}

// // 1-2. grad_query wrt query
// __global__
// void kernel_grad_query_grad_query(int N, float *grad_query, 
//                                   const float *grad_grad_query, 
//                                   const float *grad_output, 
//                                   const float *query, const float *feature,
//                                   int3 grid_sizes, int D, 
//                                   float3 min, float3 max, 
//                                   bool boundary_check) {
//   auto Gy0 = grid_sizes.y;
//   auto Gz0 = grid_sizes.z;
//   auto stride_x = Gy0 * Gz0 * D;
//   auto stride_y = Gz0 * D;
//   auto stride_z = D;
//   auto grid_sizes1 = to_float3(grid_sizes) - 1.f;
  
//   NBLA_CUDA_KERNEL_LOOP(n, N) {

//     auto b = n / D;
//     auto d = n - b * D;

//     auto querys = *(float3*)(query + b * 3);
  
//     // continuous point   
//     auto scales = grid_sizes1 / (max - min);
//     auto xyz = (querys - min) * scales;

//     // discrete points
//     auto xyz0 = floorf(xyz);    
//     xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
//     xyz0 = fminf(xyz0, grid_sizes1);
//     auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);

//     // coefficients
//     auto pqr0 = xyz1 - xyz;
//     auto pqr1 = 1.f - pqr0;

//     // scalars
//     uint x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;
//     uint x1 = xyz1.x, y1 = xyz1.y, z1 = xyz1.z;
//     auto p0 = pqr0.x, q0 = pqr0.y, r0 = pqr0.z;
//     auto p1 = pqr1.x, q1 = pqr1.y, r1 = pqr1.z;
//     auto sx = scales.x, sy = scales.y, sz = scales.z;

//     // grid features
//     auto feature_index = [&](const uint x, const uint y, const uint z) {
//       return x * stride_x + y * stride_y + z * stride_z + d;
//     };
    
//     auto f000 = feature[feature_index(x0, y0, z0)];
//     auto f001 = feature[feature_index(x0, y0, z1)];
//     auto f010 = feature[feature_index(x0, y1, z0)];
//     auto f011 = feature[feature_index(x0, y1, z1)];
//     auto f100 = feature[feature_index(x1, y0, z0)];
//     auto f101 = feature[feature_index(x1, y0, z1)];
//     auto f110 = feature[feature_index(x1, y1, z0)];
//     auto f111 = feature[feature_index(x1, y1, z1)];


//     // gradients
//     auto gg_xyz = *(float3*)(grad_grad_query + b * 3);
//     auto ggx = gg_xyz.x;
//     auto ggy = gg_xyz.y;
//     auto ggz = gg_xyz.z;
//     auto go = grad_output[n];
    
//     auto ti = go * sy * sz 
//       * (p0 * (f000 - f001 - f010 + f011) + p1 * (f100 - f101 - f110 + f111));
//     auto tj = go * sx * sz
//       * (q0 * (f000 - f001 - f100 + f101) + q1 * (f010 - f011 - f110 + f111));
//     auto tk = go * sx * sy
//       * (r0 * (f000 - f010 - f100 + f110) + r1 * (f001 - f011 - f101 + f111));

//     auto gx = ggy * tk + ggz * tj;
//     auto gy = ggz * ti + ggx * tk;
//     auto gz = ggx * tj + ggy * ti;

//     atomicAdd(grad_query + b * 3, gx);
//     atomicAdd(grad_query + b * 3 + 1, gy);
//     atomicAdd(grad_query + b * 3 + 2, gz);
//   }
// }


// void grad_query_grad_query(int N, int64_t grad_query_ptr, 
//                            int64_t grad_grad_query_ptr, 
//                            int64_t grad_output_ptr, 
//                            int64_t query_ptr, int64_t feature_ptr, 
//                            std::vector<int> grid_sizes, int D, 
//                            std::vector<float> min, std::vector<float> max, 
//                            bool boundary_check, bool accum) {
//   auto grad_query_buff = reinterpret_cast<float*>(grad_query_ptr);
//   auto grad_grad_query_buff = reinterpret_cast<float*>(grad_grad_query_ptr);
//   auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
//   auto query_buff = reinterpret_cast<float*>(query_ptr);
//   auto feature_buff = reinterpret_cast<float*>(feature_ptr);

//   NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_query_grad_query, N, 
//                                  grad_query_buff,
//                                  grad_grad_query_buff,
//                                  grad_output_buff, 
//                                  query_buff, feature_buff, 
//                                  make_int3(grid_sizes[0], grid_sizes[1], grid_sizes[2]), D, 
//                                  make_float3(min[0], min[1], min[2]),
//                                  make_float3(max[0], max[1], max[2]),
//                                  boundary_check);
// }
  
// 1-3. grad_query wrt feature
template<int window_size = 2>
__global__
void kernel_grad_query_grad_feature(int N, float *grad_feature, 
                                    const float *grad_grad_query, 
                                    const float *grad_output, 
                                    const float *query, 
                                    int G0, float growth_factor, int T0, int L, int D, 
                                    float3 min, float3 max, 
                                    bool boundary_check) {
  NBLA_CUDA_KERNEL_LOOP(n, N) { // N = L * (B * R * N)
    // query
    auto B = N / L;
    auto lb = flat_to_2d(n, B);
    auto l = lb.x, b = lb.y;
    auto querys = *(float3*)(query + b * 3);

    // variable feature hash table (T_l, D)
    auto G = compute_grid_size(G0, growth_factor, l);
    auto T = compute_table_size(G, T0);
    auto grid_sizes1 = make_float3(G, G, G) - 1.f;
    auto gx1 = grid_sizes1.x, gy1 = grid_sizes1.y, gz1 = grid_sizes1.z;

    // continuous point
    auto scales = grid_sizes1 / (max - min);
    auto sx = scales.x, sy = scales.y, sz = scales.z;
    auto xyz = (querys - min) * scales;
    auto x = xyz.x, y = xyz.y, z = xyz.z;

    // discrete points
    auto xyz0 = floorf(xyz);
    auto x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;

    // gradients
    auto gg_xyz = *(float3*)(grad_grad_query + b * 3);
    auto ggx = gg_xyz.x;
    auto ggy = gg_xyz.y;
    auto ggz = gg_xyz.z;

    auto num_params = compute_num_params(G0, growth_factor, T0, l, D);
    auto grad_feature_l = grad_feature + num_params;

    auto feature_index = [&](const uint x, const uint y, const uint z, const uint d) {
      auto hindex = hash(x, y, z, T);
      return hindex * D + d;
    };    

    auto compute_grad = [&](const float ograd, const uint x, const uint y, const uint z, const uint d, 
                            const float a, const float b, const float c) {
      auto f_idx = feature_index(x, y, z, d);
      atomicAdd(grad_feature_l + f_idx, ograd * (ggx * sx * a + ggy * sy * b + ggz * sz * c));
    };

    for (int d = 0; d < D; ++d) {
      auto o_idx = d * (L * B) + l * (B) + b;
      auto ograd = grad_output[o_idx];

      auto w = window_size;

      for (auto i = -w + 1; i <= w; ++i) {
        auto xi = clamp(x0 + i, 0.f, gx1);
        auto ci = lanczos(x - xi, w);
        auto gci = grad_coefficient(x - xi, w);

        for (auto j = -w + 1; j <= w; ++j) {
          auto yj = clamp(y0 + j, 0.f, gy1);
          auto cj = lanczos(y - yj, w);
          auto gcj = grad_coefficient(y - yj, w);

          for (auto k = -w + 1; k <= w; ++k) {
            auto zk = clamp(z0 + k, 0.f, gz1);
            auto ck = lanczos(z - zk, w);
            auto gck = grad_coefficient(z - zk, w);
            compute_grad(ograd, xi, yj, zk, d, 
                         gci * cj * ck, ci * gcj * ck, ci * cj * gck);
          }
        }
      }
    }
  }

}

void grad_query_grad_feature(int N, int64_t grad_feature_ptr, 
                             int64_t grad_grad_query_ptr, 
                             int64_t grad_output_ptr, 
                             int64_t query_ptr, 
                             int G0, float growth_factor, int T0, int L, int D, 
                             std::vector<float> min, std::vector<float> max, 
                             bool boundary_check, bool accum) {
  auto grad_feature_buff = reinterpret_cast<float*>(grad_feature_ptr);
  auto grad_grad_query_buff = reinterpret_cast<float*>(grad_grad_query_ptr);
  auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
  auto query_buff = reinterpret_cast<float*>(query_ptr);

  NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_query_grad_feature, N, 
                                 grad_feature_buff,
                                 grad_grad_query_buff,
                                 grad_output_buff, 
                                 query_buff, 
                                 G0, growth_factor, T0, L, D,
                                 make_float3(min[0], min[1], min[2]),
                                 make_float3(max[0], max[1], max[2]),
                                 boundary_check);
}

// // 2-1. grad_feature wrt grad_output
// template<bool accum = false>
// __global__
// void kernel_grad_feature_grad_grad_output(int N, float *grad_grad_output, 
//                                           const float *grad_grad_feature, 
//                                           const float *query,
//                                           int3 grid_sizes, int D, 
//                                           float3 min, float3 max, 
//                                           bool boundary_check) {
//   auto Gy0 = grid_sizes.y;
//   auto Gz0 = grid_sizes.z;
//   auto stride_x = Gy0 * Gz0 * D;
//   auto stride_y = Gz0 * D;
//   auto stride_z = D;
//   auto grid_sizes1 = to_float3(grid_sizes) - 1.f;
  
//   NBLA_CUDA_KERNEL_LOOP(n, N) {
//     auto b = n / D;
//     auto d = n - b * D;

//     auto querys = *(float3*)(query + b * 3);
  
//     // continuous point   
//     auto scales = grid_sizes1 / (max - min);
//     auto xyz = (querys - min) * scales;

//     // discrete points
//     auto xyz0 = floorf(xyz);    
//     xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
//     xyz0 = fminf(xyz0, grid_sizes1);
//     auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);

//     // coefficients
//     auto pqr0 = xyz1 - xyz;
//     auto pqr1 = 1.f - pqr0;

//     // scalars
//     uint x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;
//     uint x1 = xyz1.x, y1 = xyz1.y, z1 = xyz1.z;
//     auto p0 = pqr0.x, q0 = pqr0.y, r0 = pqr0.z;
//     auto p1 = pqr1.x, q1 = pqr1.y, r1 = pqr1.z;
    
//     // grid features
//     auto feature_index = [&](const uint x, const uint y, const uint z) {
//       return (x * stride_x) + (y * stride_y) + (z * stride_z + d);
//     };
    
//     auto ggf000 = grad_grad_feature[feature_index(x0, y0, z0)];
//     auto ggf001 = grad_grad_feature[feature_index(x0, y0, z1)];
//     auto ggf010 = grad_grad_feature[feature_index(x0, y1, z0)];
//     auto ggf011 = grad_grad_feature[feature_index(x0, y1, z1)];
//     auto ggf100 = grad_grad_feature[feature_index(x1, y0, z0)];
//     auto ggf101 = grad_grad_feature[feature_index(x1, y0, z1)];
//     auto ggf110 = grad_grad_feature[feature_index(x1, y1, z0)];
//     auto ggf111 = grad_grad_feature[feature_index(x1, y1, z1)];
    
//     // linear interpolation
//     auto ggo = p0 * q0 * r0 * ggf000
//       + p0 * q0 * r1 * ggf001
//       + p0 * q1 * r0 * ggf010
//       + p0 * q1 * r1 * ggf011
//       + p1 * q0 * r0 * ggf100
//       + p1 * q0 * r1 * ggf101
//       + p1 * q1 * r0 * ggf110
//       + p1 * q1 * r1 * ggf111;

//     grad_grad_output[n] = accum ? grad_grad_output[n] + ggo : ggo;
//   }
// }

// void grad_feature_grad_grad_output(int N, int64_t grad_grad_output_ptr, 
//                                    int64_t grad_grad_feature_ptr, 
//                                    int64_t query_ptr, 
//                                    std::vector<int> grid_sizes, int D, 
//                                    std::vector<float> min, std::vector<float> max, 
//                                    bool boundary_check, bool accum) {
//   auto grad_grad_output_buff = reinterpret_cast<float*>(grad_grad_output_ptr);
//   auto grad_grad_feature_buff = reinterpret_cast<float*>(grad_grad_feature_ptr);
//   auto query_buff = reinterpret_cast<float*>(query_ptr);

//   auto kernel = accum 
//     ? kernel_grad_feature_grad_grad_output<true> 
//     : kernel_grad_feature_grad_grad_output<false>;
//   NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel, N, 
//                                  grad_grad_output_buff,
//                                  grad_grad_feature_buff,
//                                  query_buff, 
//                                  make_int3(grid_sizes[0], grid_sizes[1], grid_sizes[2]), D, 
//                                  make_float3(min[0], min[1], min[2]),
//                                  make_float3(max[0], max[1], max[2]),
//                                  boundary_check);
// }

// // 2-2. grad_feature wrt query
// __global__
// void kernel_grad_feature_grad_query(int N, float *grad_query, 
//                                     const float *grad_grad_feature, 
//                                     const float *grad_output, 
//                                     const float *query,
//                                     int3 grid_sizes, int D, 
//                                     float3 min, float3 max, 
//                                     bool boundary_check) {
//   auto Gy0 = grid_sizes.y;
//   auto Gz0 = grid_sizes.z;
//   auto stride_x = Gy0 * Gz0 * D;
//   auto stride_y = Gz0 * D;
//   auto stride_z = D;
//   auto grid_sizes1 = to_float3(grid_sizes) - 1.f;
  
//   NBLA_CUDA_KERNEL_LOOP(n, N) {
//     auto b = n / D;
//     auto d = n - b * D;

//     auto querys = *(float3*)(query + b * 3);
  
//     // continuous point   
//     auto scales = grid_sizes1 / (max - min);
//     auto xyz = (querys - min) * scales;

//     // discrete points
//     auto xyz0 = floorf(xyz);    
//     xyz0 = fmaxf(xyz0, make_float3(0.f, 0.f, 0.f));
//     xyz0 = fminf(xyz0, grid_sizes1);
//     auto xyz1 = fminf(xyz0 + 1.f, grid_sizes1);

//     // coefficients
//     auto pqr0 = xyz1 - xyz;
//     auto pqr1 = 1.f - pqr0;

//     // scalars
//     uint x0 = xyz0.x, y0 = xyz0.y, z0 = xyz0.z;
//     uint x1 = xyz1.x, y1 = xyz1.y, z1 = xyz1.z;
//     auto p0 = pqr0.x, q0 = pqr0.y, r0 = pqr0.z;
//     auto p1 = pqr1.x, q1 = pqr1.y, r1 = pqr1.z;
//     auto sx = scales.x, sy = scales.y, sz = scales.z;

//     // grid features
//     auto feature_index = [&](const uint x, const uint y, const uint z) {
//       return x * stride_x + y * stride_y + z * stride_z + d;
//     };
    
//     auto ggf000 = grad_grad_feature[feature_index(x0, y0, z0)];
//     auto ggf001 = grad_grad_feature[feature_index(x0, y0, z1)];
//     auto ggf010 = grad_grad_feature[feature_index(x0, y1, z0)];
//     auto ggf011 = grad_grad_feature[feature_index(x0, y1, z1)];
//     auto ggf100 = grad_grad_feature[feature_index(x1, y0, z0)];
//     auto ggf101 = grad_grad_feature[feature_index(x1, y0, z1)];
//     auto ggf110 = grad_grad_feature[feature_index(x1, y1, z0)];
//     auto ggf111 = grad_grad_feature[feature_index(x1, y1, z1)];

//     // gradients
//     auto ograd = grad_output[n];
//     auto compute_grad = [&](float scale, float a0, float a1, float b0, float b1, 
//                             float d00, float d01, float d10, float d11) {
//       return ograd * scale * (a0 * b0 * d00 + a0 * b1 * d01 + a1 * b0 * d10 + a1 * b1 * d11);
//     };
//     auto gx = compute_grad(sx, q0, q1, r0, r1, 
//                            (ggf100 - ggf000), 
//                            (ggf101 - ggf001), 
//                            (ggf110 - ggf010), 
//                            (ggf111 - ggf011));
//     auto gy = compute_grad(sy, p0, p1, r0, r1, 
//                            (ggf010 - ggf000), 
//                            (ggf011 - ggf001), 
//                            (ggf110 - ggf100), 
//                            (ggf111 - ggf101));
//     auto gz = compute_grad(sz, p0, p1, q0, q1, 
//                            (ggf001 - ggf000), 
//                            (ggf011 - ggf010), 
//                            (ggf101 - ggf100), 
//                            (ggf111 - ggf110));
//     atomicAdd(grad_query + b * 3, gx);
//     atomicAdd(grad_query + b * 3 + 1, gy);
//     atomicAdd(grad_query + b * 3 + 2, gz);
//   }  
// }


// void grad_feature_grad_query(int N, int64_t grad_query_ptr, 
//                              int64_t grad_grad_feature_ptr, 
//                              int64_t grad_output_ptr, 
//                              int64_t query_ptr, 
//                              std::vector<int> grid_sizes, int D, 
//                              std::vector<float> min, std::vector<float> max, 
//                              bool boundary_check, bool accum) {
//   auto grad_query_buff = reinterpret_cast<float*>(grad_query_ptr);
//   auto grad_grad_feature_buff = reinterpret_cast<float*>(grad_grad_feature_ptr);
//   auto grad_output_buff = reinterpret_cast<float*>(grad_output_ptr);
//   auto query_buff = reinterpret_cast<float*>(query_ptr);

//   NBLA_CUDA_LAUNCH_KERNEL_SIMPLE(kernel_grad_feature_grad_query, N, 
//                                  grad_query_buff,
//                                  grad_grad_feature_buff,
//                                  grad_output_buff, 
//                                  query_buff, 
//                                  make_int3(grid_sizes[0], grid_sizes[1], grid_sizes[2]), D, 
//                                  make_float3(min[0], min[1], min[2]),
//                                  make_float3(max[0], max[1], max[2]),
//                                  boundary_check);
// }

} // voxel_hash_feature

PYBIND11_MODULE(lanczos_voxel_hash_feature_cuda, m) {
  m.doc() = "Interpolation by query on voxel hash";

  m.def("hash_index", &voxel_hash_feature::hash_index, "");

  // forward
  m.def("voxel_hash_feature", &voxel_hash_feature::voxel_hash_feature, "");

  // 1st-order gradient
  m.def("grad_query", &voxel_hash_feature::grad_query, "");
  m.def("grad_feature", &voxel_hash_feature::grad_feature, "");

  // 2nd-order gradient of 1st-order gradient wrt query
  m.def("grad_query_grad_grad_output", 
        &voxel_hash_feature::grad_query_grad_grad_output, "");
  // m.def("grad_query_grad_query", 
  //       &voxel_hash_feature::grad_query_grad_query, "");
  m.def("grad_query_grad_feature", 
        &voxel_hash_feature::grad_query_grad_feature, "");

  // // 2nd-order gradient of 1st-order gradient wrt feature
  // m.def("grad_feature_grad_grad_output", 
  //       &voxel_hash_feature::grad_feature_grad_grad_output, "");
  // m.def("grad_feature_grad_query", 
  //       &voxel_hash_feature::grad_feature_grad_query, "");
}
